
#include <hip/hip_runtime.h>
/*
 * =====================================================================================
 *
 *       Filename:  glsltest_cuda.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016年08月11日 18時15分02秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */




__global__
void moveVAO_cuda_kernel(float* v)
{
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	if(ty == 2)
	{
		int i = tx * 3 + ty;
		v[i] += 0.025;
		v[i] = ( v[i] > 1.0 ) ? -1.0f : v[i];
	}
}



void moveVAO_cuda(float* v, int vc)
{
	dim3 g(1,1,1);
	dim3 b(vc, 3, 1);
	moveVAO_cuda_kernel<<<g, b>>>(v);
}
