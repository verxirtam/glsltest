
#include <hip/hip_runtime.h>
/*
 * =====================================================================================
 *
 *       Filename:  glsltest_cuda.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016年08月11日 18時15分02秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */




__global__
void moveVAO_cuda_kernel(float* v)
{
	unsigned int tx = threadIdx.x;
	
	int i = tx * 3 + 2;
	v[i] += 0.025;
	v[i] = ( v[i] > 1.0 ) ? -1.0f : v[i];
}



void moveVAO_cuda(float* v, int vc)
{
	moveVAO_cuda_kernel<<<1, vc>>>(v);
}
